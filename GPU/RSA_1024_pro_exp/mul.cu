#include "hip/hip_runtime.h"
#include "types.h"
#include "typesgpu.h"
#include "texture.h"
#include "utils.h"
#include "print.h"

#define NTASKS 2

#define __mulmod( c0, c1, a0, a1, b0, b1, pc ) \
	asm( "{\n\t"					\
	".reg .u32 dhi;" ".reg .u32 dlo;"		\
	".reg .u32 tx;" ".reg .u32 ty;" ".reg .u32 tz;"	\
	"mul.lo.u32     %0, %2, %4;"			\
	"mul.hi.u32     %1, %2, %4;"			\
	"mad.lo.cc.u32  %1, %3, %4, %1;"		\
	"madc.hi.u32    dlo, %3, %4, 0;"		\
	"mad.lo.cc.u32  %1, %2, %5, %1;"		\
	"madc.hi.cc.u32 dlo, %2, %5, dlo;"		\
	"madc.hi.u32    dhi, %3, %5, 0;"		\
	"mad.lo.cc.u32  dlo, %3, %5, dlo;"		\
	"addc.u32       dhi, dhi, 0;"			\
	"mul.lo.u32     tx, %6, dlo;"			\
	"mul.hi.u32     ty, %6, dlo;"			\
	"mad.lo.cc.u32  ty, %6, dhi, ty;"		\
	"madc.hi.u32    tz, %6, dhi, 0;"		\
	"add.cc.u32     %0, %0, tx;"			\
	"addc.cc.u32    %1, %1, ty;"			\
	"addc.u32       dlo, tz, 0;"			\
	"mad.lo.cc.u32  %0, dlo, %6, %0;"		\
	"addc.u32       %1, %1, 0;"			\
	"}"						\
	: "=r" (c0), "=r" (c1)			\
	: "r" (a0), "r" (a1),			\
	"r" (b0), "r" (b1), "r" (pc) )
	
#define __submod( c_hi, c_lo, a_hi, a_lo, b_hi, b_lo, pc ) \
	asm( "{\n\t"				\
	".reg .s32 t;"				\
	"sub.cc.u32 %1, %5, %3;" 		\
	"subc.cc.u32 %0, %4, %2;"		\
	"subc.u32 t, 0, 0;"			\
	"slct.u32.s32 t, 0, %6, t;"		\
	"sub.cc.u32 %1, %1, t;" 		\
	"subc.u32 %0, %0, 0;"			\
	"}"					\
	: "=r" (c_hi), "=r" (c_lo)		\
	: "r" (a_hi), "r" (a_lo), "r" (b_hi), "r" (b_lo), "r" (pc) )
	
#define __addmod( c0, c1, a0, a1, b0, b1, pc ) \
	asm( "{\n\t"				\
	".reg .u32     t;"			\
	"add.cc.u32    %0, %2, %4;" 		\
	"addc.cc.u32   %1, %3, %5;"		\
	"addc.u32 t,   0, 0;"			\
	"mad.lo.cc.u32 %0, %6, t, %0;"		\
	"addc.u32      %1, %1, 0;" 		\
	"}"					\
	: "=r" (c0), "=r" (c1)		\
	: "r" (a0), "r" (a1), "r" (b0), "r" (b1), "r" (pc) )

#define __mask( c0, a0, b0) \
	asm( "{\n\t"				\
	".reg .pred  p, q;"			\
	"setp.eq.u32 p|q, %1, %2;"		\
	"@p mov.u32  %0, -1;"			\
	"@q mov.u32  %0, 0;"			\
	"}"					\
	: "=r" (c0)				\
	: "r" (a0), "r" (b0) )
	
#define __and(c0, c1, a0, a1, b0) \
	asm( "{\n\t"				\
	"and.b32 %0, %2, %4;"			\
	"and.b32 %1, %3, %4;"			\
	"}"					\
	: "=r" (c0), "=r" (c1)			\
	: "r" (a0), "r" (a1), "r" (b0) )
	
#define __xor(c0, c1, a0, a1, b0, b1) \
	asm( "{\n\t"				\
	"xor.b32 %0, %2, %4;"			\
	"xor.b32 %1, %3, %5;"			\
	"}"					\
	: "=r" (c0), "=r" (c1)			\
	: "r" (a0), "r" (a1), "r" (b0), "r" (b1) )

__device__ void static inline mm(uint2 *out, const uint2 a, const uint2 b, const uint pc)
{
	__mulmod(out->x, out->y, a.x, a.y, b.x, b.y, pc);
}

__device__ void static inline submod(uint2 *out, const uint2 a, const uint2 b, const uint pc)
{
	__submod(out->y, out->x, a.y, a.x, b.y, b.x, pc);
}

__device__ void static inline gand(uint2 *out, const uint2 a, const uint b)
{
	__and(out->y, out->x, a.y, a.x, b);
}

__device__ void static inline gxor(uint2 *out, const uint2 a, const uint2 b)
{
	__xor(out->y, out->x, a.y, a.x, b.y, b.x);
}

__device__ __forceinline__ void warpsReduceInt(volatile uint2 *m, uint tid, uint pc)
{
	__addmod(m[tid].x, m[tid].y, m[tid].x, m[tid].y, m[tid+16].x, m[tid+16].y, pc);
	
	__addmod(m[tid].x, m[tid].y, m[tid].x, m[tid].y, m[tid+8].x, m[tid+8].y, pc);
	
	__addmod(m[tid].x, m[tid].y, m[tid].x, m[tid].y, m[tid+4].x, m[tid+4].y, pc);
	
	__addmod(m[tid].x, m[tid].y, m[tid].x, m[tid].y, m[tid+2].x, m[tid+2].y, pc);
	
	__addmod(m[tid].x, m[tid].y, m[tid].x, m[tid].y, m[tid+1].x, m[tid+1].y, pc);
}

__device__ __forceinline__ void warpsReduceFlo(volatile float *v, uint tid)
{
	v[tid] += v[tid + 16];
	v[tid] += v[tid + 8];
	v[tid] += v[tid + 4];
	v[tid] += v[tid + 2];
	v[tid] += v[tid + 1];
}

__device__ void mulmod(uint2* __restrict__ c, uint2 *d, uint2 a, uint2 b, uint2* __restrict__ spinv, uint2* __restrict__ spplp, 
		       uint* __restrict__ spc, float* __restrict__ sdelta, uint bid) 
{
	__shared__ float alphas[38]; 
	uint tid = threadIdx.x;
	uint2 z, g;
	uint ci;
	
	ci = spc[blockIdx.x];
	
	mm(&z, a, b, spc[tid]);
	
	mm(&g, z, spinv[tid], spc[tid]);
	
	mm(&d[tid], g, spplp[tid], ci);
	
	warpsReduceInt(d, tid, ci);
	
	alphas[tid] = sdelta[tid];
	alphas[tid] += ((float)(g.y >> 18) / 0x4000);
	
	warpsReduceFlo(alphas, tid);
	
	if(tid < 1) 
	{
		if(alphas[0] >= 1)
			submod(&d[tid], fetch_tex((int)(alphas[0]-1) * NR + blockIdx.x), d[tid], ci);
	}
	
	if(tid < 1) { c[bid] = d[0]; }
	
	//__syncthreads();
}

__device__ void inline gpusync(int flag, volatile int* __restrict__ in, volatile int* __restrict__ out)
{
	uint tid = threadIdx.x;
	uint bid = blockIdx.x;
	int old = -9999;
	
	if(tid < 1){ in[bid] = flag; }
	
	if(bid < 1)
	{
		while(old != flag) { old = in[tid]; }
		
		__syncthreads();
		
		out[tid] = flag;
	}
	
	if(bid > 0)
	{
		if(tid < 1) { while(old != flag) { old = out[bid]; } }
	}
	
	__syncthreads();
}

__device__ __forceinline__ void linearpassing(uint2 *aux, uint *mask, volatile uint2 *gpcomp, const uint index, const uint tid)
{
	uint i;
	uint2 t;
	
	__mask(mask[tid], index, tid);
	
	aux->x = 0;
	aux->y = 0;
	
	#pragma unroll 16
	for(i = 0; i < SW; i++)
	{
		gand(&t, *(gpcomp+i*NR+tid), mask[i]);
		
		gxor(aux, t, *aux);
	}
}

__global__ void expmod(uint2* __restrict__ c, uint2* __restrict__ a, uint2* __restrict__ b, uint2* __restrict__ pcomp, 
		       volatile ushort* __restrict__ w, uint2* __restrict__ gpinv, uint2* __restrict__ pplp, 
		       uint* __restrict__ pc, float* __restrict__ delta, uint nw, int *vals1, int *vals2) 
{
	__shared__ uint2 d[38];
	__shared__ uint2 spinv[NR];
	__shared__ uint2 spplp[NR];
	__shared__ uint spc[NR];
	__shared__ float sdelta[NR];
	__shared__ uint mask[NR];
	
	uint2 aux;
	uint tid = threadIdx.x;
	uint bid = blockIdx.x;
	uint i, k, s;
	int j;
	uint2 *t1, *t2;
	uint f;
	
	spinv[tid] = gpinv[tid];
	spplp[tid] = pplp[NR * bid + tid];
	spc[tid] = pc[tid];
	sdelta[tid] = delta[tid];
	
	for(i = 0; i < TESTS; i++)
	{
		k = i * NR; // a
		s = SW * k; // pcomp
		
		if(bid < 1)
		{ pcomp[s+tid] = a[k+tid]; }
		gpusync(-1, vals1, vals2);
		
		#pragma unroll 16
		for(j = 1; j < SW; j++)
		{
			t1 = pcomp+s+j*NR;
			t2 = pcomp+s+(j-1)*NR;
			
			mulmod(t1, d, t2[tid], a[k+tid], spinv, spplp, spc, sdelta, bid);
			gpusync(j, vals1, vals2);
		}
		
		j = nw;
		
		if(bid < 1)
		{
			s = SW * k + w[j] * NR;
			b[tid] = pcomp[s+tid];
		}
		
		gpusync(j, vals1, vals2);
		j--;
		
		t1 = c+k;
		t2 = b;
		f = 1;
		
		while(j >= 0)
		{
			#pragma unroll 4
			for(s = 0; s < NB; s++)
			{
				mulmod(t1, d, t2[tid], t2[tid], spinv, spplp, spc, sdelta, bid);
				gpusync(s, vals1, vals2);
				
				if(f)
				{
					t1 = b;
					t2 = c+k;
					f = 0;
				}
				else
				{
					t1 = c+k;
					t2 = b;
					f = 1;
				}
			}
			
			s = SW * k;
			linearpassing(&aux, mask, pcomp+s, w[j], tid);
			
			mulmod(t1, d, t2[tid], aux, spinv, spplp, spc, sdelta, bid);
			gpusync(TW, vals1, vals2);
			
			if(f)
			{
				t1 = b;
				t2 = c+k;
				f = 0;
			}
			else
			{
				t1 = c+k;
				t2 = b;
				f = 1;
			}
			
			j--;
		}
		
		if(f)
		{
			if(bid < 1) { c[k+tid] = b[tid]; }
		}
	}
}

inline void convert_l2i2_a(uint2 *out, eltr_v *in, uint n, uint m)
{
	uint i,j,k = 0;
	
	for(i = 0; i < n; i++)
	{
		for(j = 0; j < m; j++)
		{
			out[k].x = in[i].x[j]; // Low
			out[k].y = in[i].x[j] >> 32; // High
			k++;
		}
	}
}

inline void convert_l2i2_b(uint2 *out, eltr in, uint n)
{
	uint i, k = 0;
	
	for(i = 0; i < n; i++)
	{
		out[k].x = in[i]; // Low
		out[k].y = in[i] >> 32; // High
		k++;
	}
}

inline void copy_pc(uint *h_pc, uint *pc, uint n)
{
	int i;
	
	for(i = n; i--; )
	{
		h_pc[i] = pc[i];
	}
}

inline void copy_delta(float *h_delta, float *delta, uint n)
{
	int i;
	
	for(i = n; i--; )
	{
		h_delta[i] = delta[i];
	}
}

inline void copy_w(ushort *h_w, ushort *w, uint n)
{
	int i;
	
	for(i = n; i--; )
	{
		h_w[i] = w[i];
	}
}

void fprint_vecintr(geltx_v in[], char chain[], uint n, uint m)
{
	uint i, j;
	FILE *pf;
	char tmp[20];
	
	sprintf(tmp, "%s.txt", chain);
	
	pf = fopen(tmp, "w");
	
	fprintf(pf, "%s:= [\n", chain);
	for(i=0; i < n-1; i++)
	{
		for(j = 0; j < m-1; j++)
		{
			fprintf(pf, "0x%.8X%.8X,", in[i].x[j].y, in[i].x[j].x);
		}
		
		fprintf(pf, "0x%.8X%.8X", in[i].x[j].y, in[i].x[j].x);
		fprintf(pf, ",\n");
	}
	
	for(j = 0; j < m-1; j++)
	{
		fprintf(pf, "0x%.8X%.8X,", in[i].x[j].y, in[i].x[j].x);
	}
	
	fprintf(pf, "0x%.8X%.8X", in[i].x[j].y, in[i].x[j].x);
	fprintf(pf, "\n];\n\n");
	
	fclose(pf);
}

void mul_gpu(eltr_v g_c[], eltr_v g_a[], eltr_v g_b[], rns_v *rns)
{
	// Variables in CPU
	uint i;
	
	// Time on GPU
	float msec = 0;
	hipEvent_t start;
	hipEvent_t stop;
	
	// Variables in CPU
	uint2 *h_a;
	uint2 *h_b;
	uint2 *h_c;
	uint2 *h_pinv;
	uint2 *h_pplp;
	uint2 *h_alphaPlp;
	float *h_delta;
	ushort *h_w;
	uint *h_pc;
	short *h_nw;
	
	// Variables in GPU
	uint2 *d_a, *d_b;
	uint2 *d_aux;
	uint2 *d_c;
	uint2 *d_pinv;
	uint *d_pc;
	uint2 *d_pplp;
	uint2 *d_alphaPlp;
	float *d_delta;
	uint2 *d_pcomp;
	ushort *d_w;
	int *d_vals1;
	int *d_vals2;
	
	hipSetDevice(1);

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipStream_t stream[NTASKS];
	
	// We allocate memory in GPU
	hipMalloc(&d_a, (TESTS * NR) * sizeof(uint2));
	hipMalloc(&d_b, (TESTS * NR) * sizeof(uint2));
	hipMalloc(&d_aux, (NR * NTASKS) * sizeof(uint2));
	hipMalloc(&d_c, (TESTS * NR * NTASKS) * sizeof(uint2));
	hipMalloc(&d_pc, (NR * NTASKS) * sizeof(uint));
	hipMalloc(&d_pinv, (NR * NTASKS) * sizeof(uint2));
	hipMalloc(&d_pplp, (NR * NR * NTASKS) * sizeof(uint2));
	hipMalloc(&d_alphaPlp, (NR * NR * NTASKS) * sizeof(uint2));
	hipMalloc(&d_delta, (NDELTA * NTASKS) * sizeof(float));
	hipMalloc(&d_pcomp, (TESTS * NR * SW * NTASKS) * sizeof(uint2));
	hipMalloc(&d_w, (TW * NTASKS) * sizeof(ushort));
	hipMalloc(&d_vals1, (NR * NTASKS) * sizeof(int));
	hipMalloc(&d_vals2, (NR * NTASKS) * sizeof(int));
	
	// The vector c is set to zero
	hipMemset(d_c, 0, (TESTS * NR * NTASKS) * sizeof(uint2));
	
	hipHostMalloc(&h_a, (TESTS * NR) * sizeof(uint2));
	hipHostMalloc(&h_b, (TESTS * NR) * sizeof(uint2));
	hipHostMalloc(&h_c, (TESTS * NR * NTASKS) * sizeof(uint2)); 
	hipHostMalloc(&h_pc, NR * sizeof(uint));
	hipHostMalloc(&h_pinv, NR * sizeof(uint2));
	hipHostMalloc(&h_pplp, (NR * NR) * sizeof(uint2));
	hipHostMalloc(&h_alphaPlp, (NR * NR) * sizeof(uint2));
	hipHostMalloc(&h_delta, NDELTA * sizeof(float));
	hipHostMalloc(&h_w, TW * sizeof(ushort));
	hipHostMalloc(&h_nw, 2 * sizeof(ushort));
	
	convert_l2i2_a(h_a, g_a, TESTS, NR);
	convert_l2i2_a(h_b, g_b, TESTS, NR);
	convert_l2i2_b(h_pinv, rns->modinvspp, NR);
	convert_l2i2_a(h_pplp, rns->pplp, NR, NR);
	convert_l2i2_a(h_alphaPlp, rns->alphaPlp, NR, NR);
	copy_pc(h_pc, rns->c, NR);
	copy_delta(h_delta, rns->delta, NR);
	copy_w(h_w, rns->w, rns->nw+1);
	h_nw[0] = h_nw[1] = rns->nw;
	
	//Create 2 streams
	for (i = 0; i < NTASKS; i++) 
	{
		hipStreamCreate(&stream[i]);
	}
	
	// Changing L1 cache configuration to 48KB - L1 and 16KB - Shared Memory 
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(expmod), hipFuncCachePreferNone);
// 	hipFuncSetCacheConfig(reinterpret_cast<const void*>(expmod), hipFuncCachePreferShared);
// 	hipFuncSetCacheConfig(reinterpret_cast<const void*>(expmod), hipFuncCachePreferL1);
	
	// Stream 0
	hipMemcpyAsync(d_a, h_a, (TESTS * NR) * sizeof(uint2), hipMemcpyHostToDevice, stream[0]);
	hipMemcpyAsync(d_w, h_w, (rns->nw + 1) * sizeof(ushort), hipMemcpyHostToDevice, stream[0]);
	hipMemcpyAsync(d_pc, h_pc, NR * sizeof(uint), hipMemcpyHostToDevice, stream[0]);
	hipMemcpyAsync(d_pinv, h_pinv, NR * sizeof(uint2), hipMemcpyHostToDevice, stream[0]);
	hipMemcpyAsync(d_pplp, h_pplp, (NR * NR) * sizeof(uint2), hipMemcpyHostToDevice, stream[0]);
	hipMemcpyAsync(d_delta, h_delta, NDELTA * sizeof(float), hipMemcpyHostToDevice, stream[0]);
	hipMemcpyAsync(d_alphaPlp, h_alphaPlp, (NR * NR) * sizeof(uint2), hipMemcpyHostToDevice, stream[0]);
	
	// Stream 1
	hipMemcpyAsync(d_b, h_b, (TESTS * NR) * sizeof(uint2), hipMemcpyHostToDevice, stream[1]);
	hipMemcpyAsync(d_w + TW, h_w, (rns->nw + 1) * sizeof(ushort), hipMemcpyHostToDevice, stream[1]);
	hipMemcpyAsync(d_pc + NR, h_pc, NR * sizeof(uint), hipMemcpyHostToDevice, stream[1]);
	hipMemcpyAsync(d_pinv + NR, h_pinv, NR * sizeof(uint2), hipMemcpyHostToDevice, stream[1]);
	hipMemcpyAsync(d_pplp + NR * NR, h_pplp, (NR * NR) * sizeof(uint2), hipMemcpyHostToDevice, stream[1]);
	hipMemcpyAsync(d_delta + NDELTA, h_delta, NDELTA * sizeof(float), hipMemcpyHostToDevice, stream[1]);
	hipMemcpyAsync(d_alphaPlp + NR * NR, h_alphaPlp, (NR * NR) * sizeof(uint2), hipMemcpyHostToDevice, stream[1]);
	
	bind_tex(d_alphaPlp);
	
	// beginning time
	hipEventRecord(start, 0);
	
	for(i = 0; i < NTASKS; i++) 
	{
		hipStreamSynchronize(stream[i]);
	}
	
	expmod<<<19, 19, 0, stream[0]>>>(d_c, d_a, d_aux, d_pcomp, d_w, d_pinv, d_pplp, d_pc, d_delta, h_nw[0], d_vals1, d_vals2); 
	expmod<<<19, 19, 0, stream[1]>>>(d_c + TESTS * NR, d_b, d_aux + NR, d_pcomp + TESTS * NR * SW, d_w + TW, d_pinv + NR, d_pplp + NR * NR, d_pc + NR, d_delta + NDELTA, h_nw[1], d_vals1 + NR, d_vals2 + NR);
	
	for(i = 0; i < NTASKS; i++) 
	{
		hipStreamSynchronize(stream[i]);
	}
	
	// End time
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&msec, start, stop);
	
	unbind_tex(d_alphaPlp);
	
// 	// Average time		Total time		All in milliseconds
	printf("%.16f\t\t%.16f\n", msec / TESTS, msec);
	
	hipMemcpyAsync(h_c, d_c, (TESTS * NR) * sizeof(uint2), hipMemcpyDeviceToHost, stream[0]);
	hipMemcpyAsync(h_c + TESTS * NR, d_c + TESTS * NR, (TESTS * NR) * sizeof(uint2), hipMemcpyDeviceToHost, stream[1]);
	
	for(i = 0; i < NTASKS; i++) 
	{
		hipStreamSynchronize(stream[i]);
	}
	
	printf("w0:= [\n");
	for(i=0; i < NR-1; i++)
	{
		printf("0x%.8X%.8X, ", h_c[i].y, h_c[i].x);
	}
	printf("0x%.8X%.8X];\n", h_c[i].y, h_c[i].x);
	
	printf("w1:= [\n");
	for(i=TESTS*NR; i < TESTS*NR+NR-1; i++)
	{
		printf("0x%.8X%.8X, ", h_c[i].y, h_c[i].x);
	}
	printf("0x%.8X%.8X];\n", h_c[i].y, h_c[i].x);
	
	for (i = 0; i < NTASKS; i++) 
	{
		hipStreamDestroy(stream[i]);
	}
	
	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	hipFree(d_pc);
	hipFree(d_pinv);
	hipFree(d_pplp);
	hipFree(d_alphaPlp);
	hipFree(d_delta);
	hipFree(d_pcomp);
	hipFree(d_w);
	hipFree(d_vals1);
	hipFree(d_vals2);
	
	hipFree(h_a);
	hipFree(h_b);
	hipFree(h_c);
	hipFree(h_pc);
	hipFree(h_pinv);
	hipFree(h_pplp);
	hipFree(h_alphaPlp);
	hipFree(h_delta);
	hipFree(h_w);
	hipFree(h_nw);
}
